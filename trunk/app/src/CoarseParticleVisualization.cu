#include "hip/hip_runtime.h"
/*******************************************************************************
Copyright (c) 2010, Steve Lesser
Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1) Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2)Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3) The name of contributors may not be used to endorse or promote products
derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL STEVE LESSER BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*******************************************************************************/

/**
@file CoarseParticleVisualization.cu
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string>
#include <iostream>
#include <sstream>
#include <time.h>
#include <math.h>
//#include <iostream>
//#include <sstream>
#include <GL/glew.h>
#include <GL/glut.h>
#include <GL/glu.h>

// includes, CUDA
#include <cuda_gl_interop.h>
#include <cutil_inline.h>
#include <cutil_gl_inline.h>
#include <cutil_math.h>
#include "ocuutil/timer.h"

// includes, CUFIRE
#include "XMLParser.h"
#include "CoarseParticleEngine.h"
#include "Projection.h"
#include "3DNavierStokes.h"
#include "Bitmap.h"
#include "fluidsGL.h"

////////////////////////////////////////////////////////////////////////////////
// constants / global variables
unsigned int window_width = 512;
unsigned int window_height = 512;

using namespace cufire;

int enableCoarseVisualization;

int numRenderTargets = 6;
enum RenderTarget
{
  RenderTexture = 0,
  RenderDensity,
  RenderFuel,
  RenderTemperature,
  RenderVelocity,
  RenderCoarseEngine
};
std::string currentRenderTargetString = "Coarse Engine";

int currentRenderTarget = RenderCoarseEngine;
CoarseParticleEngine* pEngine;
OrthographicProjection* pProjection;

float sTimestep;
float currentTime = 0;

float* d_sliceMassOutput;
float* d_sliceFuelOutput;
float2* d_sliceVelocityOutput;

float* h_sliceMassOutput;
float* h_sliceFuelOutput;
float2* h_sliceVelocityOutput;
int numSliceBytes, numSliceVelocityBytes;
int2 slicePixelDims;
float imageSize;
//angle of rotation
float xpos = 32, ypos = 32, zpos = 90, xrot = 0, yrot = 0, angle=0.0;
float3 cameraTarget, cameraUp;
float theta, phi, cameraDistance;


// rendering callbacks
void display();
void reshape(int w, int h);

float randomNormalizedFloat()
{
  return (float(rand()) / RAND_MAX);
}

float randomFloatInRange(float minVal, float maxVal)
{
  float range = maxVal - minVal;
  float returnVal = minVal + randomNormalizedFloat() * range;
  if (rand()%2==0) 
    returnVal *= -1.0f;
  return returnVal;
}

void updateSimulation(float dt)
{
  currentTime += dt;
  printf("\n\n     NEW TIME STEP\n");
  // move VBO from OpenGL context to CUDA context
  pEngine->enableCUDAVbo();
  // update coarse simulation particles
  pEngine->advanceSimulation(dt);

  // project coarse particles onto slices
  CPUTimer projectionTimer;
  projectionTimer.start();
  pProjection->setParticles(pEngine->getParticleBegins(), pEngine->getNumParticles());

  BitmapWriter massImage(slicePixelDims.x,slicePixelDims.y);
  BitmapWriter fuelImage(slicePixelDims.x,slicePixelDims.y);
  BitmapWriter velocityImage(slicePixelDims.x,slicePixelDims.y);

  for (int i = 0; i < 1; i++)
  {
    float zIntercept = 32.f;
    cutilSafeCall(hipMemset(d_sliceMassOutput,0,numSliceBytes));
    cutilSafeCall(hipMemset(d_sliceFuelOutput,0,numSliceBytes));
    cutilSafeCall(hipMemset(d_sliceVelocityOutput,0,numSliceVelocityBytes));
    // perform actual projection for slice # i
    pProjection->execute(zIntercept, d_sliceMassOutput, d_sliceFuelOutput, d_sliceVelocityOutput);
    // copy output as image
    cutilSafeCall(hipMemcpy(h_sliceMassOutput, d_sliceMassOutput, numSliceBytes, hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(h_sliceFuelOutput, d_sliceFuelOutput, numSliceBytes, hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(h_sliceVelocityOutput, d_sliceVelocityOutput, numSliceVelocityBytes, hipMemcpyDeviceToHost));

    CPUTimer fluid2DTimer;
    fluid2DTimer.start();
    replaceVelocityField(d_sliceVelocityOutput);
    dissipateDensity(dt);
    dissipateFuel(dt);
    coolTemperature(dt);
    contributeSlices(d_sliceMassOutput, d_sliceFuelOutput);
    simulateFluids(dt);
    addTextureDetail(currentTime, zIntercept);
    enforveVelocityIncompressibility(dt);
    fluid2DTimer.stop();
    printf("fluid 2D time: %f\n", fluid2DTimer.elapsed_sec());
  }

  projectionTimer.stop();
  printf("Projection time: %f\n", projectionTimer.elapsed_sec());

  // move VBO back to OpenGL
  pEngine->disableCUDAVbo();
}

void drawCube(float3 lowerLeftFront, float3 upperRightBack)
{
  glEnable(GL_BLEND);
  glBlendFunc (GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
  glBegin(GL_QUADS);
  // front face
  glColor4f(1,0,0,0.1f);
  glVertex3f(lowerLeftFront.x,lowerLeftFront.y,lowerLeftFront.z);
  glVertex3f(lowerLeftFront.x,upperRightBack.y,lowerLeftFront.z);
  glVertex3f(upperRightBack.x,upperRightBack.y,lowerLeftFront.z);
  glVertex3f(upperRightBack.x,lowerLeftFront.y,lowerLeftFront.z);
  // back face
  glVertex3f(lowerLeftFront.x,lowerLeftFront.y,upperRightBack.z);
  glVertex3f(lowerLeftFront.x,upperRightBack.y,upperRightBack.z);
  glVertex3f(upperRightBack.x,upperRightBack.y,upperRightBack.z);
  glVertex3f(upperRightBack.x,lowerLeftFront.y,upperRightBack.z);
  // left face
  glColor4f(0,1,0,0.1f);
  glVertex3f(lowerLeftFront.x,lowerLeftFront.y,lowerLeftFront.z);
  glVertex3f(lowerLeftFront.x,lowerLeftFront.y,upperRightBack.z);
  glVertex3f(lowerLeftFront.x,upperRightBack.y,upperRightBack.z);
  glVertex3f(lowerLeftFront.x,upperRightBack.y,lowerLeftFront.z);
  // right face
  glVertex3f(upperRightBack.x,lowerLeftFront.y,lowerLeftFront.z);
  glVertex3f(upperRightBack.x,lowerLeftFront.y,upperRightBack.z);
  glVertex3f(upperRightBack.x,upperRightBack.y,upperRightBack.z);
  glVertex3f(upperRightBack.x,upperRightBack.y,lowerLeftFront.z);
  // bottom face
  glColor4f(0,0,1,0.1f);
  glVertex3f(upperRightBack.x,upperRightBack.y,upperRightBack.z);
  glVertex3f(lowerLeftFront.x,upperRightBack.y,upperRightBack.z);
  glVertex3f(lowerLeftFront.x,upperRightBack.y,lowerLeftFront.z);
  glVertex3f(upperRightBack.x,upperRightBack.y,lowerLeftFront.z);
  // top face
  glVertex3f(upperRightBack.x,lowerLeftFront.y,upperRightBack.z);
  glVertex3f(lowerLeftFront.x,lowerLeftFront.y,upperRightBack.z);
  glVertex3f(lowerLeftFront.x,lowerLeftFront.y,lowerLeftFront.z);
  glVertex3f(upperRightBack.x,lowerLeftFront.y,lowerLeftFront.z);

  glEnd();
}

// found at http://www-course.cs.york.ac.uk/cgv/OpenGL/L23b.html
void DrawText(GLint x, GLint y, char* s, GLfloat r, GLfloat g, GLfloat b)
{
  int lines;
  char* p;
  glMatrixMode(GL_PROJECTION);
  glPushMatrix();
  glLoadIdentity();
  glOrtho(0.0, glutGet(GLUT_WINDOW_WIDTH), 
    0.0, glutGet(GLUT_WINDOW_HEIGHT), -1.0, 1.0);
  glMatrixMode(GL_MODELVIEW);
  glPushMatrix();
  glLoadIdentity();
  glColor3f(r,g,b);
  glRasterPos2i(x, y);
  for(p = s, lines = 0; *p; p++) {
    if (*p == '\n') {
      lines++;
      glRasterPos2i(x, y-(lines*18));
    }
    glutBitmapCharacter(GLUT_BITMAP_HELVETICA_18, *p);
  }
  glPopMatrix();
  glMatrixMode(GL_PROJECTION);
  glPopMatrix();
  glMatrixMode(GL_MODELVIEW);
}

#define M_PI 3.14159265f
void display()
{
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
  glMatrixMode(GL_MODELVIEW);
  glLoadIdentity();
  float radianTheta = theta * M_PI / 180.f;
  float radianPhi = phi * M_PI / 180.f;
  float x = cameraTarget.x + cameraDistance * sinf(radianTheta) * cosf(radianPhi);
  float y = cameraTarget.y + cameraDistance * sinf(radianTheta) * sinf(radianPhi);
  float z = cameraTarget.z + cameraDistance * cosf(radianTheta);
  gluLookAt(x,y,z, 
    cameraTarget.x,cameraTarget.y,cameraTarget.z, 
    cameraUp.x,cameraUp.y,cameraUp.z);

  // draw bounding box
  drawCube(make_float3(0,0,0), make_float3(64,64,64));
  pEngine->render();
}

void reshape(int w, int h)
{
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
  window_width = w;
  window_height = h;
  // viewport
  glMatrixMode(GL_PROJECTION);
  glViewport(0, 0, window_width, window_height);
  // projection
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  //glOrtho(-40, 40, -40, 40, 1, 80);

  gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 15.0, 200.0);
}

void timer2(int value)
{
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
  // update coarse particle simulation
  updateSimulation(sTimestep);
  //  RenderTexture = 0,
  //RenderDensity,
  //RenderFuel,
  //RenderTemperature,
  //RenderVelocity,
  //RenderCoarseEngine
  switch(currentRenderTarget)
  {
  case RenderTexture:
  case RenderFuel:
  case RenderDensity:
  case RenderTemperature:
  case RenderVelocity:
    displaySlice(currentRenderTarget);
    break;
  case RenderCoarseEngine:
    display();
    break;
  }

  char drawTime[128];
  sprintf(drawTime, "Time: %f",currentTime);
  char drawVisualization[128];
  sprintf(drawVisualization, "Visualization: %s",currentRenderTargetString.c_str());
  DrawText(10,10,drawTime,1,0,0);
  DrawText(10,30,drawVisualization,1,0,0);

  glutSwapBuffers();
  glutTimerFunc(0.0001f,timer2,value);
}

// based on OpenGL Camera Tutorial at http://www.swiftless.com/tutorials/opengl/camera.html
void keyboard (unsigned char key, int x, int y) {
  if (key == 'z' || key == 'x')
  {
    if (key=='z')
    {
      currentRenderTarget--;
      if (currentRenderTarget == -1)
        currentRenderTarget = numRenderTargets - 1;
    }
    if (key=='x')
    {
      currentRenderTarget = (currentRenderTarget + 1) % numRenderTargets;
    }
    switch(currentRenderTarget)
    {
    case RenderTexture: currentRenderTargetString = "Slice Texture"; break;
    case RenderFuel: currentRenderTargetString = "Slice Fuel"; break;
    case RenderDensity: currentRenderTargetString = "Slice Density"; break;
    case RenderTemperature: currentRenderTargetString = "Slice Temperature"; break;
    case RenderVelocity: currentRenderTargetString = "Slice Velocity"; break;
    case RenderCoarseEngine: currentRenderTargetString = "Coarse Engine"; break;
    }
  }
  if (key=='q')
  {
    phi += 10;
    if (phi >360) phi -= 360;
  }

  if (key=='e')
  {
    phi -= 10;
    if (phi < -360) phi += 360;
  }

  if (key=='w')
  {
    cameraDistance -= 10;
  }

  if (key=='s')
  {
    cameraDistance += 10;
  }

  if (key=='d')
  {
    theta += 10;
    if (theta >360) theta -= 360;
  }

  if (key=='a')
  {
    theta -= 10;
    if (theta < -360) theta += 360;
  }
  if (key==27)
  {
    exit(0);
  }
}

void setupProjection(int2 slicePixelDims)
{
  numSliceBytes = sizeof(float)*slicePixelDims.x*slicePixelDims.y;
  numSliceVelocityBytes = sizeof(float2)*slicePixelDims.x*slicePixelDims.y;
  // allocate host memory
  h_sliceMassOutput = (float*) malloc(numSliceBytes);
  h_sliceFuelOutput = (float*) malloc(numSliceBytes);
  h_sliceVelocityOutput = (float2*) malloc(numSliceVelocityBytes);
  // allocate device memory
  hipMalloc((void**)&d_sliceMassOutput, numSliceBytes);
  hipMemset(d_sliceMassOutput, 0, numSliceBytes);
  hipMalloc((void**)&d_sliceFuelOutput, numSliceBytes);
  hipMemset(d_sliceFuelOutput, 0, numSliceBytes);
  hipMalloc((void**)&d_sliceVelocityOutput, numSliceVelocityBytes);
  hipMemset(d_sliceVelocityOutput, 0, numSliceVelocityBytes);
}

int main(int argc, char* argv[])
{
  srand ( time(NULL) );
  // LOAD SIMULATION SETTINGS
  XMLParser settingsFile("ParticleSettings.xml");
  settingsFile.getInt("coarseVisualization",&enableCoarseVisualization);
  // location of starting particles
  settingsFile.setNewRoot("startingParticleRange");
  float range[2];
  // x range
  settingsFile.getFloat2("xRange",range);
  float2 xRange = make_float2(range[0],range[1]);
  xpos = range[0] + ((range[1]-range[0])/ 2.f);
  // y range
  settingsFile.getFloat2("yRange",range);
  float2 yRange = make_float2(range[0],range[1]);
  ypos = range[0] + ((range[1]-range[0])/ 2.f);
  // z range
  settingsFile.getFloat2("zRange",range);
  float2 zRange = make_float2(range[0],range[1]);
  zpos = range[1]*1.5f;
  settingsFile.resetRoot();
  // get number of starting particles and max number of particles
  int numStartingParticles;
  settingsFile.getInt("numStartingParticles",&numStartingParticles);
  int maxNumParticles;
  settingsFile.getInt("maxNumberParticles",&maxNumParticles);
  settingsFile.getFloat("timestep",&sTimestep);
  settingsFile.getFloat("imageSize",&imageSize);
  int jitterAmount;
  settingsFile.getFloat("cameraDistance",&cameraDistance);
  settingsFile.getInt("projectionJitterAmount",&jitterAmount);
  // get bounding box for coarse simulation
  settingsFile.setNewRoot("boundingBox");
  // x range
  settingsFile.getFloat2("xRange",range);
  float2 xBBox = make_float2(range[0],range[1]);
  xpos = range[0] + ((range[1]-range[0])/ 2.f);
  // y range
  settingsFile.getFloat2("yRange",range);
  float2 yBBox = make_float2(range[0],range[1]);
  ypos = range[0] + ((range[1]-range[0])/ 2.f);
  // z range
  settingsFile.getFloat2("zRange",range);
  float2 zBBox = make_float2(range[0],range[1]);
  zpos = range[0] + ((range[1]-range[0])/ 2.f);
  settingsFile.resetRoot();
  // utility values
  float3 gridCenter = make_float3(xBBox.x+(xBBox.y-xBBox.x)/2,
    yBBox.x+(yBBox.y-yBBox.x)/2,
    zBBox.x+(zBBox.y-zBBox.x)/2);
  cameraTarget = gridCenter;
  cameraUp = make_float3(0,1,0);
  theta=0;
  phi=0;
  float3 gridDims = make_float3(xBBox.y-xBBox.x,
    yBBox.y-yBBox.x,
    zBBox.y-zBBox.x);
  float projectionDepth = 2.0f;
  slicePixelDims = make_int2(imageSize,imageSize);
  float2 sliceWorldDims = make_float2(xBBox.y-xBBox.x,
    yBBox.y-yBBox.x);

  // enable either coarse particle visualization or slice simulation
  if (enableCoarseVisualization)
  {
    // First initialize OpenGL context, so we can properly set the GL for CUDA.
    // This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_ALPHA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    //register callbacks
    glutCreateWindow("CUDA Fire Simulation (Coarse Particle Visualization)");
    glutDisplayFunc(display);
    glutReshapeFunc(reshape);
    glutKeyboardFunc(keyboard);
    glewInit();
    glClearColor(0.5, 0.5, 0.5, 1.0);
    //set CUDA device
    cudaGLSetGLDevice(0);
  }
  else
  {
    setupSliceVisualization(argc,argv);
  }
  glutTimerFunc(0.0001f, timer2, 1);

  // add some random particles
  // first get area for random particles
  setupProjection(slicePixelDims);

  pProjection = new OrthographicProjection(gridCenter,gridDims,projectionDepth,slicePixelDims,sliceWorldDims,maxNumParticles,jitterAmount);
  // create particle engine
  pEngine = new CoarseParticleEngine(maxNumParticles,xBBox,yBBox,zBBox);
  pEngine->addRandomParticle(xRange,yRange,zRange,numStartingParticles);
  pEngine->flushParticles();

  setupSliceSimulation();
  //pSliceRefiner = new SliceRefiner(imageSize, argc, argv);
  // start rendering mainloop

  glutMainLoop();

  return 0;
}