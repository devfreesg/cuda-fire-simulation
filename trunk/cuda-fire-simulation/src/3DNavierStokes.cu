#include "hip/hip_runtime.h"
/********************************************************************************
Copyright (c) 2010, Steve Lesser
Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1) Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2)Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3) The name of contributors may not be used to endorse or promote products
derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL STEVE LESSER BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*******************************************************************************/

/**
@file 3DNavierStokes.cu
*/

#include "3DNavierStokes.h"
#include <cutil_math.h>

using namespace ocu;

void NavierStokes3D::setGridDimensions(int x, int y, int z)
{
  nx = x;
  ny = y;
  nz = z;
}

void NavierStokes3D::setParticles(float4* positions, float* xVel, float* yVel, float* zVel, int numParticles)
{
  m_positions = positions;
  m_xVel = xVel;
  m_yVel = yVel;
  m_zVel = zVel;
  m_numParticles = numParticles;
}

void NavierStokes3D::allocate_particles(Grid1DHostF &hposx, Grid1DHostF &hposy, Grid1DHostF &hposz, Grid1DHostF &hvx, Grid1DHostF &hvy, Grid1DHostF &hvz,
                                        Grid1DDeviceF &posx, Grid1DDeviceF &posy, Grid1DDeviceF &posz, Grid1DDeviceF &vx, Grid1DDeviceF &vy, Grid1DDeviceF &vz, 
                                        float xsize, float ysize, float zsize)
{
  hposx.init(m_numParticles,0);
  hposy.init(m_numParticles,0);
  hposz.init(m_numParticles,0);
  hvx.init(m_numParticles,0);
  hvy.init(m_numParticles,0);
  hvz.init(m_numParticles,0);

  posx.init(m_numParticles,0);
  posy.init(m_numParticles,0);
  posz.init(m_numParticles,0);
  vx.init(m_numParticles,0);
  vy.init(m_numParticles,0);
  vz.init(m_numParticles,0);

  for (int p=0; p < m_numParticles; p++) {
    float4 currentPos = m_positions[p];
    hposx.at(p) = currentPos.x;
    hposy.at(p) = currentPos.y;
    hposz.at(p) = currentPos.z;
  }

  posx.copy_all_data(hposx);
  posy.copy_all_data(hposy);
  posz.copy_all_data(hposz);
}

NavierStokes3D::NavierStokes3D()
{
  firstRun = true;
}

void NavierStokes3D::setupParams()
{
  params.init_grids(nx, ny, nz);
  params.hx = 1;
  params.hy = 1;
  params.hz = 1;
  BoundaryCondition closed;
  closed.type = BC_FORCED_INFLOW_VARIABLE_SLIP;
  params.flow_bc = BoundaryConditionSet(closed);
  BoundaryCondition neumann;
  neumann.type = BC_NEUMANN;
  params.temp_bc = BoundaryConditionSet(neumann);
  int i,j,k;
  params.init_temp.clear_zero();

  for (i=0; i < nx; i++)
  {
    for (j=0; j < ny; j++)
    {
      for (k=0; k < nz; k++) {
        params.init_u.at(i,j,k) = (i < nx / 2) ? -10 : 10;
        params.init_v.at(i,j,k) = 10;//(j < ny / 2) ? 10 : 10;
        params.init_w.at(i,j,k) = (k < nx / 2) ? -10 : 10;
        params.init_temp.at(i,j,k) = 0;//1;
      }
    }
  }
  params.max_divergence = 1e-3;
  if(!eqn.set_parameters(params))
  {
    printf("OpenCurrent parameters not properly set\n");
    exit(1);
  }
  allocate_particles(hposx, hposy, hposz, hvx, hvy, hvz, posx, posy, posz, vx, vy, vz, nx, ny, nz);
  
}

void NavierStokes3D::run()
{
  if (firstRun)
  {
    setupParams();
    firstRun = false;
  }

  double dt = .02;
  CPUTimer timer;
  timer.start();

  for (int i = 0; i < 5; i++)
  {
    if(!eqn.advance(dt))
    {
      printf("OpenCurrent parameters not properly set\n");
      exit(1);
    }
    // trace points
    sample_points_mac_grid_3d(vx, vy, vz, posx, posy, posz, eqn.get_u(), eqn.get_v(), eqn.get_w(), params.flow_bc, 1,1,1);
    hvx.copy_all_data(vx); hvy.copy_all_data(vy); hvz.copy_all_data(vz);
    for (int p=0; p < hvx.nx(); p++) {
      float3 curVel = make_float3(hvx.at(p),hvy.at(p),hvz.at(p));
      curVel *= 10.f;
      // forward Euler
      hposx.at(p) += curVel.x * dt;
      hposy.at(p) += curVel.y * dt;
      hposz.at(p) += curVel.z * dt;
    }
    // copy positions back to device
    posx.copy_all_data(hposx); posy.copy_all_data(hposy); posz.copy_all_data(hposz);
  }

  for (int p=0; p < m_numParticles; p++) {
    float4 currentPos = m_positions[p];
    currentPos.x = hposx.at(p);
    currentPos.y = hposy.at(p);
    currentPos.z = hposz.at(p);
    m_positions[p] = currentPos;
  }

  timer.stop();
  printf("Elapsed: %f, or %f fps\n", timer.elapsed_sec(), 100 / timer.elapsed_sec());
}

void NavierStokes3D::calculateVelocities(float dt)
{
}