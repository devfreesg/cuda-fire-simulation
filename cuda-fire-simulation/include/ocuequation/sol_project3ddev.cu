/*
 *  Copyright 2008-2009 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <cstdio>

#include "ocustorage/grid3dboundary.h"
#include "ocuequation/sol_project3d.h"


namespace ocu {


template<typename T>
bool Sol_ProjectDivergence3DDeviceStorage<T>::initialize_device_storage(
  int nx, int ny, int nz, double hx, double hy, double hz, Grid3DDevice<T> *u_val, Grid3DDevice<T> *v_val, Grid3DDevice<T> *w_val)
{
  if (!initialize_base_storage(nx,ny,nz,hx,hy,hz,u_val, v_val, w_val)) {
    printf("[ERROR] Sol_ProjectDivergence3DDevice::initialize_device_storage - failed to initialize base storage\n");
    return false;
  }

  u = u_val;
  v = v_val;
  w = w_val;

  if (!divergence.init(_nx, _ny, _nz, 1, 1, 1, u->pnx() - (_nx + 2), u->pny() - (_ny + 2), u->pnz() - (_nz + 2))) {
    printf("[ERROR] Sol_ProjectDivergence3DDevice::initialize_device_storage - failed to initialize divergence\n");
    return false;
  }

  if (!u_val->check_layout_match(divergence)) {
    printf("[ERROR] Sol_ProjectDivergence3DDevice::initialize_device_storage - divergence layout mismatch\n");
    return false;
  }

  return true;
}





template<typename T>
bool Sol_ProjectDivergence3DDevice<T>::solve(double tolerance)
{
  this->clear_error();
  double residual = 0;

  check_ok(apply_3d_mac_boundary_conditions_level1( *this->u, *this->v, *this->w,  this->bc, this->_hx, this->_hy, this->_hz), "Sol_ProjectDivergence3DDevice::solve - could not enforce boundary conditions");
  check_ok(this->divergence_solver.solve(), "Sol_ProjectDivergence3DDevice::solve - could not calculate divergence");
  check_ok(this->pressure_solver.solve(residual, tolerance, 15), "Sol_ProjectDivergence3DDevice::solve - could not solve for pressure\n");
  check_ok(this->gradient_solver.solve(), "Sol_ProjectDivergence3DDevice::solve - could not subtract gradient of pressure\n");
  check_ok(apply_3d_mac_boundary_conditions_level1( *this->u, *this->v, *this->w,  this->bc, this->_hx, this->_hy, this->_hz), "Sol_ProjectDivergence3DDevice::solve - could not enforce boundary conditions\n");

  return !this->any_error();
}

template<typename T>
bool Sol_ProjectDivergence3DDevice<T>::solve_divergence_only()
{
  this->clear_error();

  check_ok(apply_3d_mac_boundary_conditions_level1( *this->u, *this->v, *this->w,  this->bc, this->_hx, this->_hy, this->_hz), "Sol_ProjectDivergence3DDevice::solve - could not enforce boundary conditions");
  check_ok(this->divergence_solver.solve(), "Sol_ProjectDivergence3DDevice::solve - could not calculate divergence");

  return !this->any_error();
}


template<typename T>
bool Sol_ProjectDivergence3DDevice<T>::initialize_storage(
  int nx, int ny, int nz, double hx, double hy, double hz, Grid3DDevice<T> *u_val, Grid3DDevice<T> *v_val, Grid3DDevice<T> *w_val)
{
  if (!initialize_base_storage(nx,ny,nz,hx,hy,hz,u_val, v_val, w_val)) {
    printf("[ERROR] Sol_ProjectDivergence3DDevice::initialize_storage - failed to initialize base storage\n");
    return false;
  }

  if (!initialize_device_storage(nx,ny,nz,hx,hy,hz,u_val, v_val, w_val)) {
    printf("[ERROR] Sol_ProjectDivergence3DDevice::initialize_storage - failed to initialize device storage\n");
    return false;
  }

  this->pressure_solver.bc.xneg = convert_bc_to_poisson_eqn(this->bc.xneg);
  this->pressure_solver.bc.xpos = convert_bc_to_poisson_eqn(this->bc.xpos);
  this->pressure_solver.bc.yneg = convert_bc_to_poisson_eqn(this->bc.yneg);
  this->pressure_solver.bc.ypos = convert_bc_to_poisson_eqn(this->bc.ypos);
  this->pressure_solver.bc.zneg = convert_bc_to_poisson_eqn(this->bc.zneg);
  this->pressure_solver.bc.zpos = convert_bc_to_poisson_eqn(this->bc.zpos);

  if (!divergence_solver.initialize_storage(nx, ny, nz, hx, hy, hz, this->u, this->v, this->w, &this->divergence)) {
    printf("[ERROR] Sol_ProjectDivergence3DDevice::initialize_storage - failed to initialize divergence_solver\n");
    return false;
  }

  if (!pressure_solver.initialize_storage(nx, ny, nz, hx, hy, hz, &this->divergence)) {
    printf("[ERROR] Sol_ProjectDivergence3DDevice::initialize_storage - failed to initialize pressure_solver\n");
    return false;
  }

  if (!gradient_solver.initialize_storage(nx, ny, nz, hx, hy, hz, this->u, this->v, this->w, &this->pressure_solver.pressure())) {
    printf("[ERROR] Sol_ProjectDivergence3DDevice::initialize_storage - failed to initialize gradient_solver\n");
    return false;
  }
  this->gradient_solver.coefficient = -1;

  if (!u_val->check_layout_match(this->pressure_solver.pressure())) {
    printf("[ERROR] Sol_ProjectDivergence3DDevice::initialize_storage - pressure layout mismatch\n");
    return false;
  }

  this->pressure_solver.pressure().clear_zero();
  this->pressure_solver.convergence = CONVERGENCE_LINF;

  return true;
}



template class Sol_ProjectDivergence3DDeviceStorage<float>;
template class Sol_ProjectDivergence3DDevice<float>;

#ifdef OCU_DOUBLESUPPORT
template class Sol_ProjectDivergence3DDeviceStorage<double>;
template class Sol_ProjectDivergence3DDevice<double>;
#endif // OCU_DOUBLESUPPORT



}

