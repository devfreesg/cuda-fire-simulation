/*
 *  Copyright 2008-2009 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "hip/hip_runtime.h"
#include <cstdio>

#include "ocustorage/grid3d.h"
#include "ocuutil/timer.h"
#include "ocuutil/timing_pool.h"

namespace ocu {


template<typename T>
bool 
Grid3DHost<T>::copy_all_data(const Grid3DDevice<T> &from)
{
  if (!this->check_layout_match(from)) {
    printf("[ERROR] Grid3DHost::copy_all_data - mismatch: (%d, %d, %d) != (%d, %d, %d)\n", this->pnx(), this->pny(), this->pnz(), from.pnx(),  from.pny(),  from.pnz());
    return false;
  }

  GPUTimer timer;
  timer.start();
  if ((unsigned int) hipSuccess != hipMemcpy(this->_buffer, from.buffer(), sizeof(T) * this->num_allocated_elements(), hipMemcpyDeviceToHost)) {
    printf("[ERROR] Grid3DHost::copy_all_data - hipMemcpy failed\n");
    return false;
  }
  timer.stop();
  global_timer_add_timing("hipMemcpy(DeviceToHost)", timer.elapsed_ms());
 
  return true;
}




template bool Grid3DHost<float>::copy_all_data(const Grid3DDevice<float> &from);
template bool Grid3DHost<int>::copy_all_data(const Grid3DDevice<int> &from);
template bool Grid3DHost<double>::copy_all_data(const Grid3DDevice<double> &from);



} // end namespace

